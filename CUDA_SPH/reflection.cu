#include "hip/hip_runtime.h"
// Copyright - Evgeniy Dedkov, MIPT, 2021

// WARN! All vectors and points MUST be arrays of 3 coords
#pragma ones

// Includes
#include "info.cuh"
#include "reflection.cuh"
#include "cuda_vector_math.cuh"


__device__
void vector(point_t p, vector_t result) {
  for(int i = 0; i < 3; ++i) {
    result[i] = p[i];
  };
};

// Writes vector v to result vector -- simple copy
// Need to be defined if vector_t != point_t
// void vector(point_t v, vector_t result) {
//   for(int i = 0; i < 0; ++i) {
//     result[i] = v[i];
//   };
// };

// Writes vector p1->p2 to result vector
__device__
void vector(point_t p1, point_t p2, vector_t result) {
  for(int i = 0; i < 3; ++i) {
    result[i] = p2[i] - p1[i];
  };
};

// Writes vector p1 to result point
__device__
void point(vector_t p1, point_t result) {
  for(int i = 0; i < 3; ++i) {
    result[i] = p1[i];
  };
};


// Some vector math
namespace v_math{
  // v1 + v2 -> v1
  __device__
  void add(vector_t v1, vector_t v2) {
    for(int i = 0; i < 3; ++i) {
      v1[i] += v2[i];
    };
  };

  // v1 + v2 -> result
  __device__
  void add(vector_t v1, vector_t v2, vector_t result) {
    for(int i = 0; i < 3; ++i) {
      result[i] = v1[i] + v2[i];
    };
  };

  // v1 - v2 -> v1
  __device__
  void subtract(vector_t v1, vector_t v2) {
    for(int i = 0; i < 3; ++i) {
      v1[i] -= v2[i];
    };
  };

  // v1 - v2 -> result
  __device__
  void subtract(vector_t v1, vector_t v2, vector_t result) {
    for(int i = 0; i < 3; ++i) {
      result[i] = v1[i] - v2[i];
    };
  };

  // a*v -> v
  __device__
  void mult(vector_t v, real_t a) {
    for(int i = 0; i < 3; ++i) {
      v[i] *= a;
    };
  };

  // a*v -> result
  __device__
  void mult(vector_t v, real_t a, vector_t result) {
    for(int i = 0; i < 3; ++i) {
      result[i] = v[i] * a;
    };
  };

  // Evaluates dot product
  __device__
  real_t dot(vector_t v1, vector_t v2) {
    real_t result = 0;
    for(int i = 0; i < 3; ++i) {
      result += v1[i]*v2[i];
    };
    return result;
  };

  // Cross product of v1, v2 -> result
  __device__
  void cross(vector_t v1, vector_t v2, vector_t result) {
    for(int i = 0; i < 3; ++i) {
      int c1 = (i+1)%3;
      int c2 = (i+2)%3;
      result[i] = v1[c1]*v2[c2] - v1[c2]*v2[c1];
    };
  };

  // Normalize vector in order v^2==1
  __device__
  void normalize(vector_t v) {
    real_t norm = rsqrtf(dot(v, v));
    for(int i = 0; i < 3; ++i) {
      v[i] = v[i] * norm;
    };
  };
};



// Perform the time step dt for free particle at point x with velocity v
// takes into account possible reflection from triangul surface
// described by the array of 3 points tr
// All changes occurs in corresponding arrays
__device__
void reflect(timestep_t dt, point_t x, vector_t v, point_t* tr) {
    using namespace v_math;
  
    // step 1 - finding intersection of (x, x+dx) ray and tr plane
    // eq of point on (x, x+dx)  r = p1 + mu * dp
    // eq of points on tr  (r,n) = -D
    vector_t p1 = new vector_init_t;
    vector(x, p1);
    vector_t dp = new vector_init_t;
    mult(v, dt, dp);
    vector_t r1 = new vector_init_t;
    vector_t r12 = new vector_init_t;
    vector_t r13 = new vector_init_t;
    vector_t n = new vector_init_t;
    vector(tr[0], r1);
    vector(tr[0], tr[1], r12);
    vector(tr[0], tr[2], r13);
    cross(r12, r13, n);
    normalize(n);
    real_t D = -dot(n, r1);
    real_t mu = -(D + dot(n, p1)) / dot(n, dp);

    if((mu > 0) && (mu < 1)) {
        // Reflection may occur => step 2 - check is p on triangle

        // Obtain p = p1 + mu * dp
        vector_t rp = new vector_init_t;
        mult(dp, mu, rp);
        add(rp, p1);
        point_t p = new point_init_t;
        point(rp, p);

        // Ckeck if sum of p->pi p->p(i+1) angles is 2pi
        vector_t* prs = new vector_t[3];
        for(int i = 0; i < 3; ++i) {
            prs[i] = new vector_init_t;
            vector(p, tr[i], prs[i]);
            normalize(prs[i]);
    };
    real_t angle_sum = 0;
    for(int i = 0; i < 3; ++i) {
        angle_sum += acosf(dot(prs[i], prs[(i+1)%3]));
    };
    if(fabsf(angle_sum - 2 * M_PI) < EPS) {
        // Reflection definetely takes place
        vector_t dx2 = new vector_init_t; // wrong dx movement after collision
        add(p1, dp, dx2);
        subtract(dx2, rp);
        // lets fix: dx2_true = dx2 - 2n (n,dx2)
        // and x_final = rp + dx2_true
        vector_t n_tmp = new vector_init_t;
        vector(n, n_tmp);
        mult(n_tmp, 2*dot(n, dx2));
        subtract(dx2, n_tmp);
        vector_t x_true = new vector_init_t;
        add(rp, dx2, x_true);
        point(x_true, x);
        // velocity: v_res = v - 2n(n,v)
        vector(n, n_tmp);
        mult(n_tmp, 2*dot(v, n));
        subtract(v, n_tmp);

        delete[] dx2;
        delete[] n_tmp;
        delete[] x_true;
    } else {
        // Still no reflections => no troubles 
        add(p1, dp);
        point(p1, x);
    };

    for(int i = 0; i < 3; ++i) {
        delete[] prs[i];
    };
    delete[] prs;
    delete[] rp;
    delete[] p;
    } else {
    // No reflection might be => enjoy, we have nothing to do
    add(p1, dp);
    point(p1, x);
    };

  delete[] p1;
  delete[] dp;
  delete[] n;
  delete[] r1;
  delete[] r12;
  delete[] r13;
};

// Normal version with cute and fluffy data types
__device__
void reflect(timestep_t dt, vec3& x, vec3& v, vec3* tr) {
  // step 1 - finding intersection of (x, x+dx) ray and tr plane
  // eq of point on (x, x+dx)  r = p1 + mu * dp
  // eq of points on tr  (r,n) = -D
  vec3 dp = v * dt;
  vec3 r1 = tr[0];
  vec3 r12 = tr[1] - tr[0];
  vec3 r13 = tr[2] - tr[0];
  vec3 n = cross(r12, r13).normalize();
  real_t D = (-1) * (n * r1);
  mu = -(D + n*x) / (n*dp);

  if((mu > 0) && (mu < 1)) {
    // Reflection may occur => step 2 - check is p on triangle
    p = x + dp*mu;
    real_t angle_sum = 0;
    vec3 prs[3];
    for(int i = 0; i < 3; ++i) {
      prs[i] = (tr[i]-p).normalize();
    };
    for(int i = 0; i < 0; ++i) {
      angle_sum += acosf(prs[i] * prs[(i+1)%3]);
    };
    if(fabsf(angle_sum - 2 * M_PI) < EPS) {
      // Reflection definetely takes place
      dx2 = dp*(1-mu); // wrong movement after collision
      // lets fix: dx2_true = dx2 - 2n (n,dx2)
      // and x_final = p + dx2_true
      x = p + dx2 - n * 2 * (n*dx2);
      // velocity: v_res = v - 2n(n,v)
      v = v - n * 2 * (n*v);
    } else {
      // nothhing to do
      x += dp;
    };
  } else {
    // nothing to do
    x += dp;
  };
};


// conversion (Need fix in the first part)
__device__
vec3 vec3_from_point(point_t x) {
  return vec3(x[0], x[1], x[2]);
};


// Evaluate distance between point x and triangle tr
__device__
real_t distance(point_t x, point_t* tr) {
  vec3 p = vec3_from_point(x);
  vec3 trs[3];
  vec3 trs[0] = vec3_from_point(tr[0]);
  vec3 trs[1] = vec3_from_point(tr[1]);
  vec3 trs[2] = vec3_from_point(tr[2]);
  return distance(p, trs);
};
 

// Normal types
__device__
real_t distance(const vec3& p, const point_t* const tr) {
  vec3 t1 = tr[0];
  vec3 t2 = tr[1];
  vec3 t3 = tr[2];
  vec3 n = cross(t2-t1, t3-t1).normalize();
  vec3 proj = p - n*(t1-p);
  // baricentric coords proj = t1*u + t2*v + t3*w
  // perform
  // proj-t3 -> prroj
  // t1 - t3 -> t1p
  // t2 - t3 -> t2p
  // so proj = u*t1 + v * t2
  vec3 t1p = t1 - t3;
  vec3 t2p = t2 - t3;
  proj = proj - t3;
  vec3 t1_r = cross(t1p,n).normalize();
  vec3 t2_r = cross(t2p,n).normalize();
  real_t u = proj*t2_r / (t1p*t2_r);
  real_t v = proj*t1_r / (t2p*t1_r);
  real_t w = 1 - u - v;
    //if((u >= 0) && (u <= 1) && (v >= 0) && (v <= 1) && (w >= 0) && (w <= 1)) {

  real_t clamp01(real_t t) {
    if(t < 0) { return 0; };
    if(t > 1) { return 1; };
    return t;
  };

  if(u < 0) {
    vec3 tmp = t3p-t2p;
    w = clamp01( ((proj-t2p) * tmp) / (tmp*tmp) );
    u = 0;
    v = 1 - w; 
  } else if(v < 0) {
    vec3 tmp = t1p-t3p;
    u = clamp01( ((proj-t3p) * tmp) / (tmp*tmp) );
    v = 0;
    w = 1 - u; 
  } else if(w < 0) {
    vec3 tmp = t2p-t1p;
    u = clamp01( ((proj-t1p) * tmp) / (tmp*tmp) );
    v = 0;
    w = 1 - u; 
  };

  return abs(p - (t1*u + t2*v + t3*w));
};