#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "neighbour.cuh"

/*
__device__ Neighbour::Neighbour(size_t n)
{
	nei_numbers = (size_t*)malloc(n * sizeof(size_t));
	for (size_t i = 0; i < n; ++i)
		nei_numbers[i] = 0;

}
*/
Neighbour::Neighbour(size_t _n)
{
	n = _n;

	hipMalloc((void**)&d_this->nei_numbers, n * sizeof(size_t));
	hipMalloc((void**)&d_this->neighbour, n * n * sizeof(size_t));

	hipMalloc((void**)&d_this, sizeof(Neighbour));
	hipMemcpy(d_this, this, sizeof(Neighbour), hipMemcpyHostToDevice);
}

__device__ size_t Neighbour::NeigboursNumber(size_t i)
{
	return nei_numbers[i];
}

__device__ size_t* Neighbour::getNeighbours(size_t i)
{
	return neighbour + n*i;
}

__global__ void initNeighbour(Particle* pts, Neighbour* nei)
{
	return;
}
