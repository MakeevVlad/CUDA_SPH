#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include "sph.cuh"
#include "particle.cuh"
#include "utils.cuh"
//#include "neighbour.cuh"



__global__ void test(Particle* p)
{
	printf("%f %f %f", p[0].pos[0], p[0].pos[1], p[0].pos[2]);
}

__global__ void tester1()
{
	printf("Tester:\n");
	/*
	printf("%f\n", pts[2].pos[0]);
	printf("%d\n", nei->n);
	for (size_t i = 0; i < nei->n; ++i)
	{
		for (size_t j = 0; j < nei->nei_numbers[i]; ++j)
			printf("%d ", nei->neighbour[nei->n * i + j]);
		printf("\n");
	}*/

}


int main()
{

	Particle* ps = new Particle[N];
	/*
	for (size_t i = 0; i <5; ++i)
		for (size_t j = 0; j < 5; ++j)
		{
			ps[i * 5 + j].set_pos(i + 0.1, j + 4.1, 0);
			ps[i * 5 + j].set_vel(0, 0, 0);
			ps[i * 5 + j].set_ax(0, 0, 0);
		}
		*/
	ps[0].set_pos(0.5, 0, 0);
	ps[0].set_vel(0, 0, 0);
	ps[0].set_ax(0, 0, 0);
	ps[1].set_pos(-0.5, 0, 0);
	ps[1].set_vel(0, 0, 0);
	ps[1].set_ax(0, 0, 0);
	float dt = 0.01;
	size_t iterations = 10000;
	size_t pts_number = N;
	solver(ps, dt, iterations, pts_number);


	return 0; 
}
