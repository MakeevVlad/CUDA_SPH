#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>

#include "sph.cuh"
#include "particle.cuh"
#include "utils.cuh"
//#include "neighbour.cuh"



__global__ void test(Particle* p)
{
	printf("%f %f %f", p[0].pos[0], p[0].pos[1], p[0].pos[2]);
}

__global__ void tester1()
{
	printf("Tester:\n");
	/*
	printf("%f\n", pts[2].pos[0]);
	printf("%d\n", nei->n);
	for (size_t i = 0; i < nei->n; ++i)
	{
		for (size_t j = 0; j < nei->nei_numbers[i]; ++j)
			printf("%d ", nei->neighbour[nei->n * i + j]);
		printf("\n");
	}*/

}


int main()
{

	Particle* ps = new Particle[N];
	int p = sqrt(N);
	for (size_t i = 0; i <p; ++i)
		for (size_t j = 0; j < p; ++j)
		{
			ps[i * p + j].set_pos(i*1.1 - p/2 , j * 1.1 - p / 2, 0);
			ps[i * p + j].set_vel(0, 0, 0);
			ps[i * p + j].set_ax(0, 0, 0);
		}
		

	float dt = 0.001;
	size_t iterations = 10000;
	size_t pts_number = N;
	solver(ps, dt, iterations, pts_number);


	return 0; 
}
